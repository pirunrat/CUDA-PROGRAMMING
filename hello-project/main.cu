#include <hip/hip_runtime.h>
#include <iostream>

__global__ void hello_from_gpu() {
    printf("Hello from GPU!\n");
}

int main() {
    hello_from_gpu<<<1, 1>>>();
    hipDeviceSynchronize(); // Wait for the kernel to finish
    return 0;
}
