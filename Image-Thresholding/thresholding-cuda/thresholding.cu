#include <hip/hip_runtime.h>
#include <iostream>

__global__ void threshold_kernel(unsigned char* d_img, int size, unsigned char thresh) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_img[idx] = (d_img[idx] > thresh) ? 255 : 0;
    }
}

// Mark for export on Windows
#ifdef _WIN32
#define EXPORT __declspec(dllexport)
#else
#define EXPORT
#endif

extern "C" EXPORT void threshold_image(unsigned char* img, int size, unsigned char thresh) {
    unsigned char *d_img;
    hipMalloc(&d_img, size);
    hipMemcpy(d_img, img, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    threshold_kernel<<<gridSize, blockSize>>>(d_img, size, thresh);
    hipDeviceSynchronize();

    hipMemcpy(img, d_img, size, hipMemcpyDeviceToHost);
    hipFree(d_img);
}
